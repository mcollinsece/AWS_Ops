
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel function to print "Hello, World!" from GPU
__global__ void helloFromGPU() {
    printf("Hello, World! from GPU thread %d\n", threadIdx.x);
}

int main() {
    // Print from CPU
    std::cout << "Hello, World! from CPU\n";

    // Launch the kernel with one block and one thread
    helloFromGPU<<<1, 10>>>();

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    return 0;
}